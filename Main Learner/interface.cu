#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "interface.cuh"

int size_of_float = sizeof(float);

float blockSize = 64.0;

float h_zero = 0;
float h_one = 1.0;

__device__ float e = 2.71828182845904523536028747135266;
__device__ float zero = 0;
__device__ float half = 0.5;
__device__ float one = 1.0;
__device__ float two = 2.0;
__device__ float four = 4.0;

__device__ float swish(float x) {
	return x / (one + pow(e, -x));
}

__device__ float dswish(float x) {
	float z = pow(e, x);

	return (z * (z + x + one)) / pow((z + one), two);
}

__device__ float relu(float x) {
	if (x > zero) {
		return x;
	}
	else {
		return zero;
	}
}

__device__ float drelu(float x) {
	if (x >= zero) {
		return one;
	}
	else {
		return zero;
	}
}

__device__ float lerelu(float x) {
	if (x >= zero) {
		return x;
	}
	else {
		return half * x;
	}
}

__device__ float dlerelu(float x) {
	if (x >= zero) {
		return one;
	}
	else {
		return half;
	}
}

__device__ float hyptan(float x) {
	float z1, z2, y;
	z1 = pow(e, x);
	z2 = pow(e, -x);

	y = (z1 - z2) / (z1 + z2);

	return y;
}

__device__ float dhyptan(float x) {
	float z = pow(e, x);

	float y = (z * four) / pow((z + one), two);

	return y;
}

__device__ float sigmoid(float x) {
	return one / (one + pow(e, -x));
}

__device__ float dsigmoid(float x) {
	float z = pow(e, x);

	return z / pow((z + one), two);
}

__device__ float softsign(float x){
	return x / (one + fabs(x));
}

__device__ float dsoftsign(float x){
	return one / pow((one + fabs(x)), two);
}

__device__ float elu(float x) {
	if(x < zero){
		return 0.1*(pow(e, x)-one);
	}
	else{
		return x;
	}
}

__device__ float delu(float x) {
	if (x >= zero){
		return one;
	}
	else{
		return 0.1*pow(e, x);
	}
}

__device__ float activate(int activation_value, float x) {
	switch (activation_value) {
		case 0:
			return swish(x);
		case 1:
			return dswish(x);
		case 2:
			return relu(x);
		case 3:
			return drelu(x);
		case 4:
			return lerelu(x);
		case 5:
			return dlerelu(x);
		case 6:
			return hyptan(x);
		case 7:
			return dhyptan(x);
		case 8:
			return sigmoid(x);
		case 9:
			return dsigmoid(x);
		case 10:
			return softsign(x);
		case 11:
			return dsoftsign(x);
		case 12:
			return elu(x);
		case 13:
			return delu(x);
		default:
			return x;
	}
}

__device__ inline void protectedAddition(float* address, float value){
    float old = value; 
    float new_old;

    do{
        new_old = atomicExch(address, 0.0f);
        new_old += old;
    }
    while ((old = atomicExch(address, new_old))!=0.0f);
};

__global__ void varyfind(float *output_values, int offset_output, float *target_values, int offset_target, float *sum, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        if(target_values[offset_target + index] != 0){
            protectedAddition(&sum[0], fabs((target_values[offset_target + index] - output_values[offset_output + index])/target_values[offset_target + index]));
        }
        else{
            protectedAddition(&sum[0], fabs(target_values[offset_target + index] - output_values[offset_output + index]));
        }
    }
}

__global__ void setValue(float *values, int offset, float value, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        values[offset + index] = value;
    }
}

__global__ void activateValue(float *values, int offset, int activation_value, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        values[offset + index] = activate(activation_value, values[offset + index]);
    }
}

__global__ void vectorMatrixMultiply(float *values, int offset_previous, int previous_count, int offset_current, int current_count, float *weight_values, int offset_weight, int weight_count, int bias_count, int inclusive_count){
    int weight_index = blockIdx.x*blockDim.x + threadIdx.x;
    int previous_index = weight_index%inclusive_count;
    int current_index = (weight_index - previous_index)/inclusive_count;

    if(current_index < current_count){
        if(weight_index < weight_count){
            if(previous_index < previous_count){
                protectedAddition(&values[offset_current + current_index], values[offset_previous + previous_index]*weight_values[offset_weight + weight_index]);
                //__syncthreads();
            }
            else{
                protectedAddition(&values[offset_current + current_index], weight_values[offset_weight + weight_index]);
                //__syncthreads();
            }
        }
    }
}

__global__ void vectorSubtract(float *target_values, int offset_target, float *output_values, int offset_output, float *derivative_sum_values, int offset_derivative, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        derivative_sum_values[offset_derivative + index] = output_values[offset_output + index] - target_values[offset_target + index];
    }
}

__global__ void vectorActivateMultiply(float *values, int offset, float *derivative_sum_values, int offset_derivative, int activation_value, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        derivative_sum_values[offset_derivative + index] *= activate(activation_value, values[offset + index]);
    }
}

__global__ void vectorMatrixAdjust(float *derivative_sum_values, int offset_derivative_next, int offset_derivative_current, float *weight_values, int offset_weight, int weight_count, float *values, int offset, int next_count, int current_count, int bias_count, int inclusive_count, float learning_rate){
    int weight_index = blockIdx.x*blockDim.x + threadIdx.x;
    int next_index = weight_index%inclusive_count;
    int current_index = (weight_index - next_index)/inclusive_count;

    if(current_index < current_count){
        if(weight_index < weight_count){
            if(next_index < next_count){
                if (offset_derivative_next >= zero) {
                    protectedAddition(&derivative_sum_values[offset_derivative_next + next_index], derivative_sum_values[offset_derivative_current + current_index]*weight_values[offset_weight + weight_index]);
                    //__syncthreads();
                }

                protectedAddition(&weight_values[offset_weight + weight_index], -(derivative_sum_values[offset_derivative_current + current_index]*values[offset + next_index]*learning_rate));
                //__syncthreads();
            }
            else{
                protectedAddition(&weight_values[offset_weight + weight_index], -(derivative_sum_values[offset_derivative_current + current_index]*learning_rate));
                //__syncthreads();
            }
        }
    }
}

__global__ void vectorCopy(float *source_values, int offset_source, float *dest_values, int offset_dest, int count){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    if(index < count){
        dest_values[offset_dest + index] = source_values[offset_source + index];
    }
}

__host__ void forward(int line_count, int line_num, int *activation_values, int *hidden_sizes, int layer_count, int bias_count, float *values, float *weight_values){
    int previous_count, current_count;

    int previous_neuron_distance, current_neuron_distance;

	previous_neuron_distance = line_num*hidden_sizes[0];
	current_neuron_distance = line_count*hidden_sizes[0];

    int weight_distance, current_weight_count;

    weight_distance = h_zero;

    int vectorSize, matrixSize;

	for (int layer_num = h_zero; layer_num < layer_count+1; layer_num++) {
        previous_count = hidden_sizes[layer_num];
		current_count = hidden_sizes[layer_num+1];

        current_weight_count = (previous_count + bias_count)*current_count;

        vectorSize = ceil(current_count/blockSize);
        matrixSize = ceil(current_weight_count/blockSize);

        setValue<<<vectorSize, blockSize>>>(values, current_neuron_distance, h_zero, current_count);
        hipDeviceSynchronize();

        vectorMatrixMultiply<<<matrixSize, blockSize>>>(values, previous_neuron_distance, previous_count, current_neuron_distance, current_count, weight_values, weight_distance, current_weight_count, bias_count, previous_count+bias_count);
        hipDeviceSynchronize();

        activateValue<<<vectorSize, blockSize>>>(values, current_neuron_distance, activation_values[layer_num], current_count);
        hipDeviceSynchronize();

        previous_neuron_distance = current_neuron_distance;
        current_neuron_distance += current_count;
        weight_distance += current_weight_count;
    }
}

__host__ void backward(int line_count, int line_num, int *activation_values, int *hidden_sizes, int layer_count, int bias_count, int hidden_count, int weight_count, float *derivative_sum_values, float *values, float *target_values, float *weight_values, float learning_rate){
	int current_count, next_count;

	int current_neuron_distance, next_neuron_distance;

	current_neuron_distance = line_count*hidden_sizes[0] + hidden_count;
	next_neuron_distance = current_neuron_distance;

	int derivative_neuron_distance;

	derivative_neuron_distance = hidden_count;

    int weight_distance, current_weight_count;

    weight_distance = weight_count;

    int vectorSize, matrixSize;

    vectorSize = ceil(hidden_sizes[layer_count+1]/blockSize);

	vectorSubtract<<<vectorSize, blockSize>>>(target_values, line_num*hidden_sizes[layer_count+1], values, current_neuron_distance, derivative_sum_values, hidden_count, hidden_sizes[layer_count+1]);
    hipDeviceSynchronize();

	for (int layer_num = layer_count+1; layer_num > h_zero; layer_num--) {
		current_count = hidden_sizes[layer_num];
		next_count = hidden_sizes[layer_num-1];

		current_weight_count = (next_count + bias_count)*current_count;
		weight_distance -= current_weight_count;

        vectorSize = ceil(current_count/blockSize);
        matrixSize = ceil(current_weight_count/blockSize);

        if(layer_num == 1){
            next_neuron_distance = line_num*hidden_sizes[0];
        }
        else{
            next_neuron_distance -= next_count;

            setValue<<<vectorSize, blockSize>>>(derivative_sum_values, derivative_neuron_distance-next_count, h_zero, next_count);
            hipDeviceSynchronize();
        }

		vectorActivateMultiply<<<vectorSize, blockSize>>>(values, current_neuron_distance, derivative_sum_values, derivative_neuron_distance, activation_values[layer_num-1]+1, current_count);
        hipDeviceSynchronize();

		vectorMatrixAdjust<<<matrixSize, blockSize>>>(derivative_sum_values, derivative_neuron_distance-next_count, derivative_neuron_distance, weight_values, weight_distance, current_weight_count, values, next_neuron_distance, next_count, current_count, bias_count, next_count+bias_count, learning_rate);
        hipDeviceSynchronize();

        current_neuron_distance -= next_count;
		derivative_neuron_distance -= next_count;
	}
}

__host__ void train(double min_diff, double learning_rate, int cycles, int line_count_train, float *input_values_train, float *target_values_train, int line_count_validate, float *input_values_validate, float *target_values_validate, int *activation_values, int *hidden_sizes, int layer_count, int bias_count, int hidden_count, int weight_count, float *weight_values) {
    int input_count = hidden_sizes[0];
    int output_count = hidden_sizes[layer_count+1];

    float *d_values_train, *d_values_validate;

    hipMalloc(&d_values_train, (line_count_train * input_count + hidden_count + output_count)*size_of_float);
    hipMalloc(&d_values_validate, (line_count_validate * input_count + hidden_count + output_count)*size_of_float);

    hipMemcpy(d_values_train, input_values_train, line_count_train*input_count*size_of_float, hipMemcpyHostToDevice);
    hipMemcpy(d_values_validate, input_values_validate, line_count_validate*input_count*size_of_float, hipMemcpyHostToDevice);

    float *d_target_values_train, *d_target_values_validate;

    hipMalloc(&d_target_values_train, (line_count_train * output_count)*size_of_float);
    hipMalloc(&d_target_values_validate, (line_count_validate * output_count)*size_of_float);

    hipMemcpy(d_target_values_train, target_values_train, (line_count_train * output_count)*size_of_float, hipMemcpyHostToDevice);
    hipMemcpy(d_target_values_validate, target_values_validate, (line_count_validate * output_count)*size_of_float, hipMemcpyHostToDevice);

    float *d_weight_values;

    hipMalloc(&d_weight_values, weight_count*size_of_float);

    hipMemcpy(d_weight_values, weight_values, weight_count*size_of_float, hipMemcpyHostToDevice);

    float *derivative_sum_values;

    hipMalloc(&derivative_sum_values, (hidden_count + output_count)*size_of_float);


    int output_offset_train = line_count_train*input_count + hidden_count;
    int output_offset_validate = line_count_validate*input_count + hidden_count;

    int target_offset;

    float *d_sum;

    hipMalloc(&d_sum, size_of_float);

    float *sum = (float*) malloc(size_of_float);
    float diff_value;

    float avg_diff_train = min_diff;
    float avg_diff_validate = min_diff;

    float *prev_diff_values = (float*) malloc(line_count_train);
    float *prev_prev_diff_values = (float*) malloc(line_count_train);

    float *learning_rate_values = (float*) malloc(line_count_train);

    memset(learning_rate_values, ((float) learning_rate), line_count_train*size_of_float);

    float learning_rate_coefficient;


    int vectorSize = ceil(output_count/blockSize);


    int cycle = 0;

    while ((cycles == -1 || cycle < cycles) && avg_diff_train >= min_diff) {
        avg_diff_train = 0;

        for (int line_num_train = h_zero; line_num_train < line_count_train; line_num_train++) {
            forward(line_count_train, line_num_train, activation_values, hidden_sizes, layer_count, bias_count, d_values_train, d_weight_values);
            backward(line_count_train, line_num_train, activation_values, hidden_sizes, layer_count, bias_count, hidden_count, weight_count, derivative_sum_values, d_values_train, d_target_values_train, d_weight_values, learning_rate_values[line_num_train]);

            hipMemset(d_sum, h_zero, size_of_float);
            target_offset = line_num_train*output_count;
            varyfind<<<vectorSize, blockSize>>>(d_values_train, output_offset_train, d_target_values_train, target_offset, d_sum, output_count);
            hipDeviceSynchronize();
            hipMemcpy(sum, d_sum, size_of_float, hipMemcpyDeviceToHost);

            diff_value = sum[0]/output_count;
            avg_diff_train += diff_value;

            learning_rate_coefficient = fabs(((prev_diff_values[line_num_train]-prev_prev_diff_values[line_num_train])/prev_prev_diff_values[line_num_train])/((diff_value-prev_diff_values[line_num_train])/prev_diff_values[line_num_train]));

            if(cycle > h_one && diff_value != prev_diff_values[line_num_train] && prev_diff_values[line_num_train] != prev_prev_diff_values[line_num_train] && learning_rate_coefficient < 1.1){
                learning_rate_values[line_num_train] *= learning_rate_coefficient;
            }
            else{
                learning_rate_values[line_num_train] = learning_rate;
            }

            prev_prev_diff_values[line_num_train] = prev_diff_values[line_num_train];
            prev_diff_values[line_num_train] = diff_value;
        }

        avg_diff_train /= line_count_train;

        avg_diff_validate = 0;

        for(int line_num_validate = h_zero; line_num_validate < line_count_validate; line_num_validate++){
            forward(line_count_validate, line_num_validate, activation_values, hidden_sizes, layer_count, bias_count, d_values_validate, d_weight_values);

            hipMemset(d_sum, h_zero, size_of_float);
            target_offset = line_num_validate*output_count;
            varyfind<<<vectorSize, blockSize>>>(d_values_validate, output_offset_validate, d_target_values_validate, target_offset, d_sum, output_count);
            hipDeviceSynchronize();
            hipMemcpy(sum, d_sum, size_of_float, hipMemcpyDeviceToHost);

            diff_value = sum[0]/output_count;
            avg_diff_validate += diff_value;
        }

        avg_diff_validate /= line_count_validate;

        printf("%.16f : %.16f\n", avg_diff_train, avg_diff_validate);

        cycle++;
    }
    printf("%.16f\n", weight_values[0]);
    hipMemcpy(weight_values, d_weight_values, weight_count*size_of_float, hipMemcpyDeviceToHost);
    printf("%.16f\n", weight_values[0]);
    hipFree(d_values_train);
    hipFree(d_values_validate);

    hipFree(d_target_values_train);
    hipFree(d_target_values_validate);
    
    hipFree(d_weight_values);

    hipFree(derivative_sum_values);

    hipFree(d_sum);

    free(prev_diff_values);
    free(prev_prev_diff_values);

    free(learning_rate_values);

    free(sum);
}


__host__ void test(int line_count, float *input_values, float *output_values, int *activation_values, int *hidden_sizes, int layer_count, int bias_count, int hidden_count, int weight_count, float *weight_values){
    int input_count = hidden_sizes[0];
    int output_count = hidden_sizes[layer_count+1];

    float *d_values;

    hipMalloc(&d_values, (line_count*input_count + hidden_count + output_count)*size_of_float);

    hipMemcpy(d_values, input_values, line_count*input_count*size_of_float, hipMemcpyHostToDevice);

    float *d_weight_values;

    hipMalloc(&d_weight_values, weight_count*size_of_float);

    hipMemcpy(d_weight_values, weight_values, weight_count*size_of_float, hipMemcpyHostToDevice);

    float *d_output_values;

    hipMalloc(&d_output_values, line_count*output_count*size_of_float);

    int vectorSize = ceil(output_count/blockSize);

    for (int line_num = h_zero; line_num < line_count; line_num++) {
        forward(line_count, line_num, activation_values, hidden_sizes, layer_count, bias_count, d_values, d_weight_values);

        vectorCopy<<<vectorSize, blockSize>>>(d_values, line_count*input_count + hidden_count, d_output_values, line_num*output_count, output_count);
        hipDeviceSynchronize();
    }

    hipMemcpy(output_values, d_output_values, line_count*output_count*size_of_float, hipMemcpyDeviceToHost);

    hipFree(d_values);
    
    hipFree(d_weight_values);

    hipFree(d_output_values);
}